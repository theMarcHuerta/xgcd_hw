#include <iostream>
#include <mutex>
#include <cstdint>
#include <numeric>
#include <map>
#include <limits>
#include <atomic>
#include <vector>
#include <cstring>
#include <hip/hip_runtime.h>

// 1) Define how big your histogram can be.
//    We've changed this to 32, as you said your max iterations won't exceed 32.
// #define MAX_ITERS 256
#define MAX_ITERS 32

// We'll store partial results in this struct on the GPU.
struct GpuThreadResult {
    // For truncate mode
    uint64_t sum_iters_trunc;
    double   sum_trunc_clears;
    int      trunc_max_iter;
    int      valid_pairs;
    // We'll skip storing the entire histogram here (we'll store it in deviceHistogram below).
};

// A struct to hold results of the device-based XGCD step
struct XgcdResultDevice {
    uint32_t gcd;
    int iterations;
    double avgBitClears;
};

__device__ 
void d_updateThreadResult(GpuThreadResult &res, const XgcdResultDevice &xg, int a, int b)
{
    // Summation
    res.sum_iters_trunc     += xg.iterations;
    res.sum_trunc_clears    += xg.avgBitClears;
    // Track max iteration
    if (xg.iterations > res.trunc_max_iter) {
        res.trunc_max_iter = xg.iterations;
    }
    // Count the pair
    res.valid_pairs++;
}

// Return the bit-length of x (0 if x==0).
__device__ int d_bit_length(uint32_t x) {
    if (x == 0) return 0;
    // Use built-in on GPU
    return 32 - __clz(x);
}

// The device version of xgcd_bitwise that doesn't use C++ STL.
__device__ XgcdResultDevice xgcd_bitwise_device(uint32_t a_in, uint32_t b_in,
                                               int total_bits, int approx_bits,
                                               bool integer_rounding /* or bool use_truncate */)
{
    // 1) Mask off any bits above total_bits
    uint32_t mask = (total_bits >= 32) ? 0xFFFFFFFF : ((1u << total_bits) - 1);
    uint32_t a = a_in & mask;
    uint32_t b = b_in & mask;

    if (b > a) {
        uint32_t temp = a; a = b; b = temp;
    }
    if (b == 0) {
        return {a, 0, 0.0};
    }
    if (a == 0) {
        return {b, 0, 0.0};
    }

    int iteration_count = 0;
    double total_bit_clears = 0.0;

    while (b != 0) {
        iteration_count++;

        // align b's MSB with a's MSB
        int len_a = d_bit_length(a);
        int len_b = d_bit_length(b);
        int shift_amount = len_a - len_b;
        uint32_t b_aligned = (shift_amount > 0) ? (b << shift_amount) : b;

        // get top bits
        auto get_fixed_top_bits = [&](uint32_t x_val) {
            if (x_val == 0) return 0u;
            int length = d_bit_length(x_val);
            if (length <= approx_bits) {
                return x_val << (approx_bits - length);
            } else {
                int shift_down = length - approx_bits;
                return x_val >> shift_down;
            }
        };

        uint32_t a_top = get_fixed_top_bits(a);
        uint32_t b_top = get_fixed_top_bits(b_aligned);

        // approximate Q
        uint64_t numerator = (uint64_t)a_top << approx_bits;
        uint32_t quotient = (b_top == 0) ? 0 : (uint32_t)(numerator / b_top);

        // ------------------------------------------------
        // STEP 5: Shift the quotient, then do Q++ (unconditional)
        // ------------------------------------------------
        uint32_t Q_pre_round = (quotient << shift_amount) >> (approx_bits - 1);
        uint32_t Q = (Q_pre_round >> 1);
        Q++;

        // compute product
        uint64_t product = (uint64_t)b * Q;

        // also compute product_two, picking whichever residual is smaller
        uint64_t product_two = (uint64_t)b * (Q_pre_round >> 1);

        uint32_t residual  = (a >= product) ? (a - (uint32_t)product)
                                            : ((uint32_t)product - a);
        uint32_t residual2 = (a >= product_two) ? (a - (uint32_t)product_two)
                                                : ((uint32_t)product_two - a);

        if (residual2 < residual) {
            residual = residual2;
            Q = (Q_pre_round >> 1);
        }

        // Count how many bits got cleared (msb_a - msb_res)
        int msb_a = d_bit_length(a);
        int msb_r = d_bit_length(residual);
        int cleared = msb_a - msb_r;
        if (cleared < 0) cleared = 0;
        total_bit_clears += (double)cleared;

        // prepare next iteration
        if (residual > b) {
            a = residual;
        } else {
            // swap
            uint32_t tmp = b;
            b = residual;
            a = tmp;
        }

        // If it's taking too long, break and indicate an error.
        // For 6-bit input, let's say 200 is already suspiciously large.
        if (iteration_count > 200) {
            // we can return a sentinel gcd or do something to show an error
            printf("INFINITE LOOP DETECTED: a_in=%u b_in=%u a=%u b=%u iteration_count=%d\n",
                   a_in, b_in, a, b, iteration_count);
            return { 0, iteration_count, 0.0 };
        }
    }

    // final iteration's bit clearing adjustment? (If needed, replicate your Python logic.)
    // We'll skip for brevity or just do something simpler:
    // total_bit_clears += d_bit_length(a);

    double avg_clears = 0.0;
    if (iteration_count > 0) {
        avg_clears = total_bit_clears / iteration_count;
    }

    return {a, iteration_count, avg_clears};
}

// A CUDA kernel that directly enumerates all (a, b) pairs for the assigned range of `a`.
// Each block processes one range chunk of `a`s in a grid-stride loop style.
__global__
void bruteForceKernel(int bits, int approx_bits,
                      bool force_a_msb,
                      bool int_rounding,
                      int a_min, int a_max, 
                      GpuThreadResult *deviceResults,
                      unsigned int *deviceHistogram)
{
    // -----------------
    // 1) Create a block-local histogram in shared memory
    // -----------------
    __shared__ unsigned int blockHist[MAX_ITERS];
    // Initialize the shared histogram to zero
    for (int i = threadIdx.x; i < MAX_ITERS; i += blockDim.x) {
        blockHist[i] = 0;
    }
    __syncthreads();

    // Each thread accumulates partial sums:
    GpuThreadResult myPartial;
    myPartial.sum_iters_trunc   = 0;
    myPartial.sum_trunc_clears  = 0.0;
    myPartial.trunc_max_iter    = -1;
    myPartial.valid_pairs       = 0;

    // Use a grid-stride loop over 'a':
    for (int a = blockIdx.x * blockDim.x + threadIdx.x + a_min;
         a <= a_max;
         a += gridDim.x * blockDim.x)
    {
        // printf("Thread %d (global %d) enumerating a=%d ...\n", 
        //        threadIdx.x, blockIdx.x*blockDim.x + threadIdx.x, a);
        for (int b = 1; b <= a; b++) {
            // CALL THE DEVICE VERSION (not the old xgcd_bitwise from xgcd_impl)
            XgcdResultDevice xg = xgcd_bitwise_device(
                a, b, bits, approx_bits, int_rounding
            );

            // accumulate partial data
            myPartial.sum_iters_trunc += xg.iterations;
            myPartial.sum_trunc_clears += xg.avgBitClears;
            if (xg.iterations > myPartial.trunc_max_iter) {
                myPartial.trunc_max_iter = xg.iterations;
            }
            myPartial.valid_pairs++;

            // -----------------
            // 2) Update the block-local histogram instead of the global one
            // -----------------
            int iters = xg.iterations;
            if (iters < MAX_ITERS) {
                atomicAdd(&blockHist[iters], 1);
            }
        }
    }

    // Sync so that all updates to blockHist are done before we aggregate
    __syncthreads();

    // Store partial sums in device memory
    int tId = blockIdx.x * blockDim.x + threadIdx.x;
    deviceResults[tId] = myPartial;

    // -----------------
    // 3) One final pass to merge blockHist into the global histogram
    //    Only do so for threads within the valid range
    // -----------------
    for (int i = threadIdx.x; i < MAX_ITERS; i += blockDim.x) {
        if (blockHist[i] > 0) {
            atomicAdd(&deviceHistogram[i], blockHist[i]);
        }
    }
}

// Host function to run GPU kernel, gather partial results, and print them.
int main(int argc, char* argv[])
{
    // ~~~ Reuse same argument parsing as CPU code ~~~
    int bits = 12;
    int approx_bits = 4;
    bool force_a_msb = false;
    bool int_rounding = true;

    // Minimal parse:
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--bits") == 0 && i + 1 < argc) {
            bits = atoi(argv[++i]);
        } else if (strcmp(argv[i], "--approx_bits") == 0 && i + 1 < argc) {
            approx_bits = atoi(argv[++i]);
        } else if (strcmp(argv[i], "--force_a_msb") == 0) {
            force_a_msb = true;
        } else if (strcmp(argv[i], "--int_rounding") == 0) {
            int_rounding = true;
        }
    }

    // Decide the range for 'a'
    int a_min, a_max;
    if (force_a_msb) {
        a_min = 1 << (bits - 1);
        a_max = (1 << bits) - 1;
    } else {
        a_min = 1;  // skipping 0
        a_max = (1 << bits) - 1;
    }
    int total_a = (a_max - a_min + 1);

    // Choose a block and grid size
    int blockSize = 256; 
    int gridSize = (total_a + blockSize - 1) / blockSize;
    // But to avoid huge grids, clamp it
    if (gridSize > 65535) gridSize = 65535;

    // Allocate device array for partial results
    int totalThreads = blockSize * gridSize;
    GpuThreadResult *deviceResults;
    hipMalloc(&deviceResults, totalThreads * sizeof(GpuThreadResult));

    // 4) Allocate device array for the global histogram
    unsigned int *deviceHistogram;
    hipMalloc(&deviceHistogram, MAX_ITERS * sizeof(unsigned int));
    // Zero out the histogram on the device
    hipMemset(deviceHistogram, 0, MAX_ITERS * sizeof(unsigned int));

    // Launch kernel (add our new deviceHistogram argument)
    bruteForceKernel<<<gridSize, blockSize>>>(
        bits, approx_bits, force_a_msb, int_rounding, 
        a_min, a_max, deviceResults,
        deviceHistogram
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    std::cout << "[DEBUG] After kernel, about to gather results..." << std::endl;

    // Copy partial results back to host
    std::vector<GpuThreadResult> hostResults(totalThreads);
    hipMemcpy(hostResults.data(), deviceResults, totalThreads * sizeof(GpuThreadResult), hipMemcpyDeviceToHost);
    hipFree(deviceResults);

    // Also copy the histogram back from GPU
    std::vector<unsigned int> hostHistogram(MAX_ITERS);
    hipMemcpy(
        hostHistogram.data(), deviceHistogram,
        MAX_ITERS * sizeof(unsigned int),
        hipMemcpyDeviceToHost
    );
    hipFree(deviceHistogram);

    // Aggregate results on the host
    uint64_t total_trunc_iters = 0;
    double   total_trunc_clears = 0.0;
    int      global_trunc_max_iter = -1;
    int      total_valid_pairs = 0;

    for (auto &res : hostResults) {
        total_trunc_iters     += res.sum_iters_trunc;
        total_trunc_clears    += res.sum_trunc_clears;
        if (res.trunc_max_iter > global_trunc_max_iter) {
            global_trunc_max_iter = res.trunc_max_iter;
        }
        total_valid_pairs     += res.valid_pairs;
    }

    // Print some final info
    std::cout << "\n===== GPU RESULTS (TRUNCATE MODE) =====\n";
    uint64_t totalPairs = (uint64_t)0;
    for(int a=a_min; a<=a_max; a++){
        totalPairs += a; 
    }
    double trunc_iter_mean = double(total_trunc_iters) / double(totalPairs);
    // As a quick hack, we interpret sum_trunc_clears as a total, but we must 
    // also consider how you originally computed average. You might refine this as needed.

    double trunc_clears_mean = (total_trunc_clears / total_trunc_iters) * trunc_iter_mean;

    std::cout << "  Mean Iterations    : " << trunc_iter_mean << "\n";
    std::cout << "  Mean Bit Clears    : " << trunc_clears_mean << "\n";
    std::cout << "  Max Iterations     : " << global_trunc_max_iter << "\n";
    std::cout << "  Valid Pairs        : " << total_valid_pairs << "\n";
    std::cout << "Tested a total of " << totalPairs << " pairs.\n";

    // 7) Print out the histogram
    std::cout << "\n===== HISTOGRAM (TRUNCATE MODE) =====\n";
    for (int i = 0; i < MAX_ITERS; i++) {
        unsigned int count = hostHistogram[i];
        if (count > 0) {
            std::cout << "Iterations " << i << ": " << count << "\n";
        }
    }

    std::cout << "\n--- End of GPU brute force test ---\n";
    return 0;
} 